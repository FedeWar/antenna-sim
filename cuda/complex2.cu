
struct complex2
{
	__device__ complex2(float xr, float xi, float yr, float yi) :
		x(make_hipComplex(xr, xi)),
		y(make_hipComplex(yr, yi))
	{}

	__device__ complex2(hipComplex _x, hipComplex _y) :
		x(_x),
		y(_y)
	{}
	
	__device__ complex2 operator+(const complex2& c) const
	{
		return complex2(hipCaddf(x, c.x), hipCaddf(y, c.y));
	}

	__device__ complex2 operator*(const float a) const
	{
		return complex2(x * a, y * a);
	}

	__device__ complex2& operator*=(const float a)
	{
		x.x *= a;
		x.y *= a;
		y.x *= a;
		y.y *= a;
		return *this;
	}

	__device__ complex2 operator*(const hipComplex& c) const
	{
		return complex2(hipCmulf(x, c), hipCmulf(y, c));
	}

	__device__ complex2& operator+=(const complex2& c)
	{
		x.x += c.x.x;
		x.y += c.x.y;
		y.x += c.y.x;
		y.y += c.y.y;

		return *this;
	}

	hipComplex x;
	hipComplex y;
};
