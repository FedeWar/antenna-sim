#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_complex.h>
#include <math_functions.h>

%DEFINES%

extern _CRTIMP __host__ __device__ __device_builtin__ float __cdecl sqrtf(float) __THROW;
extern _CRTIMP __host__ __device__ __device_builtin__ float __cdecl expf(float) __THROW;
extern _CRTIMP __host__ __device__ __device_builtin__ float __cdecl abs(float) __THROW;
extern __device__ __device_builtin__ void sincosf(float x, float *sptr, float *cptr) __THROW;

// Pycuda does not support multiple file compilations, we have to improvise
%CUFILES%

// c = k * rr1
// exp(-1j * self.k * rr1)
// c viene moltiplicato per -j
__device__ hipComplex cexp(const hipComplex& c)
{
	float a = expf(-c.y);
	float r, i;

	sincosf(-c.x, &r, &i);

	return make_hipComplex(a * r, a * i);
}

__device__ float3 add(const float3& v1, const float3& v2)
{
	return make_float3(v1.x + v2.x, v1.y + v2.y, v1.z + v2.z);
}

__device__ complex2 Ji(float3 r, float3 r1)
{
	// FIXME in teoria si possono levare
	// 1) Perché va implementato un modo più elastico
	// 2) L'integrazione è garantita tra [-10, +10)
	// filo lungo 20
	if(abs(r1.x) < 0.001 && abs(r1.y) < 10)
	{
		return complex2(1, 0, 0, 0);
	}
	else
	{
		return complex2(0, 0, 0, 0);
	}
}

__device__ hipComplex G(const float3& r, const float3& r1)
{
	// Fa in modo che il denominatore non si annulli
	const float epsilon = 0.000001f;
	const float pi = 3.1415927410125f;
	const hipComplex k = make_hipComplex(1, 0);

	const float rr1 = vabs(sub(r, r1)) + epsilon;
	const hipComplex N = cexp(k * rr1);
	const float D = 1.0f / (4 * pi * rr1);

	return N * D;
}

/*
 * La funzione da integrare lungo r1.
 *
 * /param r		Punto in cui calcolare l'integrale.
 * /param r1	Somma degli incrementi differenziali fino a ora.
 * /return		Valore della funzione.
 */
__device__ complex2 f(const float3& r, const float3& r1)
{
	return Ji(r, r1) * G(r, r1);
}

/*
 * Esegue un passo dell'algoritmo RK4.
 * 
 * /param r			Punto in cui calcolare l'integrale.
 * /param r1		Somma degli incrementi differenziali fino a ora.
 * /param A 		Valore per A trovato nel precedente step.
 * /param dr		Incremento differenziale.
 * /define abs_dr	Lunghezza dell'incremento differenziale per r1
 * /return Il valore di A incrementato di uno step.
 */
__device__ inline void step(const float3& r, const float3& r1, complex2& A, const float3& dr)
{
	#define abs_dr 0.1f
	complex2 k1 = f(r, r1);
	k1 += f(r, add(r1, mul(dr, 0.5f))) * 2.0f;	// k2
	k1 += f(r, add(r1, dr));	// k3

	A += k1 * (abs_dr / 4.0f);
}

extern "C"
__global__ void compute(float* out_points)
{
	const int x = blockIdx.x * bwidth + threadIdx.x;
	const int y = blockIdx.y * bheight + threadIdx.y;
	const int z = blockIdx.z;// * bdepth + thread.z
	const int point = (z * HEIGHT + y) * WIDTH + x;
	const float3 r = make_float3((x + offx) * SCALE, (y + offy) * SCALE, (z + offz) * SCALE);
	const float3 dr = make_float3(0, 0.1f, 0);
	const int its = 200;

	complex2 A(0, 0, 0, 0);
	float3 r1 = make_float3(0, -10, 0);

	for(int i = 0; i < its; i++)
	{
		step(r, r1, A, dr);
		r1 += dr;
	}

	out_points[point] = vabs(A.x);
}
